#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// Error macro for cuda calls
#define CHECK_CUDA_ERROR(call)                                       \
  {                                                                  \
    const hipError_t err = call;                                    \
    if (err != hipSuccess) {                                        \
      printf("CUDA Error: %s at line %d\n", hipGetErrorString(err), \
             __LINE__);                                              \
      exit(1);                                                       \
    }                                                                \
  }
#define N 12
// Since we are using flatten 2d arrays
#define IDX2D(i, j) (i * (N + 2) + j)
#define TOLERANCE 0.003f

// Globals only on the GPU

__device__ int is_done;
__device__ int debug_counter;
__device__ int arrivalLock;
__device__ int departureLock;
__device__ int count;

__device__ float *gpu_arr_a;
__device__ float *gpu_arr_b;

/********* GPU FUNCTIONS BEGIN HERE *****************/

// Cstar Lock implementation in CUDA
__device__ void Lock(int *mutex) {
  while (atomicCAS(mutex, 0, 1) != 0) { /*spinning*/
  }
}

// Cstar Unlock implementation in CUDA
__device__ void Unlock(int *mutex) { atomicExch(mutex, 0); }

__device__ bool aggregate(const bool mydone) {
  bool result;

  // Arrival phase
  Lock(&arrivalLock);
  const int old_count_add = atomicAdd(&count, 1);
  atomicAnd(&is_done, mydone ? 1 : 0);
  if (old_count_add + 1 < N) {
    Unlock(&arrivalLock);
  } else {
    Unlock(&departureLock);
  }

  // departure phase
  Lock(&departureLock);
  const int old_count_sub = atomicSub(&count, 1);
  result = (is_done != 0);
  if (old_count_sub - 1 > 0) {
    Unlock(&departureLock);
  } else {
    atomicExch(&is_done, 1);
    Unlock(&arrivalLock);
  }
  return result;
}

__global__ void jacobi_relaxation() {
  const int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
  float change = 0.0f, max_change = 0.0f;
  bool done = false;

  printf("Starting thread idx=%d\n", idx);

  do {
    max_change = 0.0f;
    for (int j = 1; j <= N; j++) {
      gpu_arr_b[IDX2D(idx, j)] =
          (gpu_arr_a[IDX2D(idx - 1, j)] + gpu_arr_a[IDX2D(idx + 1, j)] +
           gpu_arr_a[IDX2D(idx, j - 1)] + gpu_arr_a[IDX2D(idx, j + 1)]) /
          4.0f;
      change = fabsf(gpu_arr_b[IDX2D(idx, j)] - gpu_arr_a[IDX2D(idx, j)]);
      if (change > max_change) {
        max_change = change;
      }
    }
    __syncthreads();

    for (int j = 1; j <= N; j++) {
      gpu_arr_a[IDX2D(idx, j)] = gpu_arr_b[IDX2D(idx, j)];
    }
    done = aggregate(max_change < TOLERANCE);
  } while (!done);
}

/********* GPU FUNCTIONS END HERE *******************/

/********* CPU FUNCTIONS BEGIN HERE *****************/

void initMatrix(float *a) {
  for (int i = 0; i <= N + 1; i++) {
    for (int j = 0; j <= N + 1; j++) {
      a[IDX2D(i, j)] = (float)(rand() % 200) / 200.0f;
    }
  }
}

void printMatrix(const float *A) {
  for (int i = 0; i <= N + 1; i++) {
    printf("Row: %2d ->", i);
    for (int j = 0; j <= N + 1; j++) {
      printf("%.3f  ", A[IDX2D(i, j)]);
    }
    printf("\n");
  }
}

// Set-up GPU -> Launch kernel -> Verify results -> Tear-down GPU
int main(void) {
  float *cpu_arr_a, *gpu_arr_temp_a, *gpu_arr_temp_b;
  int zero = 0, one = 1;

  const size_t arr_size = (N + 2) * (N + 2) * sizeof(float);

  // Allocate in cpu
  cpu_arr_a = (float *)malloc(arr_size);
  initMatrix(cpu_arr_a);

  printf("Initial matrix:\n");
  printMatrix(cpu_arr_a);

  // Allocate in gpu
  CHECK_CUDA_ERROR(hipMalloc(&gpu_arr_temp_a, arr_size));
  CHECK_CUDA_ERROR(hipMalloc(&gpu_arr_temp_b, arr_size));

  // Copy pointers to global variables
  CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(count), &zero, sizeof(int)));
  CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(arrivalLock), &zero, sizeof(int)));
  CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(departureLock), &one, sizeof(int)));
  CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(is_done), &one, sizeof(int)));
  CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(debug_counter), &zero, sizeof(int)));

  CHECK_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(gpu_arr_a), &gpu_arr_temp_a, sizeof(float *)));
  CHECK_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(gpu_arr_b), &gpu_arr_temp_b, sizeof(float *)));

  // Copy to gpu
  CHECK_CUDA_ERROR(
      hipMemcpy(gpu_arr_temp_a, cpu_arr_a, arr_size, hipMemcpyHostToDevice));

  // Setup the blocks and call the kernel

  printf("Starting kernel execution...\n");
  jacobi_relaxation<<<1, N>>>();

  // Check for kernel launch errors
  CHECK_CUDA_ERROR(hipGetLastError());

  // Wait for the kernel to finish and check for errors
  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  // Copy from gpu to cpu
  CHECK_CUDA_ERROR(
      hipMemcpy(cpu_arr_a, gpu_arr_temp_a, arr_size, hipMemcpyDeviceToHost));

  printMatrix(cpu_arr_a);

  // CleanUp
  free(cpu_arr_a);
  CHECK_CUDA_ERROR(hipFree(gpu_arr_temp_a));
  CHECK_CUDA_ERROR(hipFree(gpu_arr_temp_b));

  // Reset the Gpu
  CHECK_CUDA_ERROR(hipDeviceReset());
  return 0;
}

/********* CPU FUNCTIONS END HERE********************/
